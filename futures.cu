#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <omp.h>

#include "CudaEvent.h"
#include "CudaStream.h"

// a = b+c
__global__
void sum(double* a, double* b, double* c, size_t N) {
    size_t tid = threadIdx.x + blockDim.x*blockIdx.x;
    size_t grid_step = blockDim.x * gridDim.x;

    while(tid<N) {
        a[tid] = b[tid] + c[tid];
        tid += grid_step;
    }
}

bool initialize_cuda() {
    hipError_t result = hipInit(0);
    return result == hipSuccess;
}

int main(void) {
    if(!initialize_cuda()) {
        std::cerr << "unable to initialize CUDA" << std::endl;
        return 1;
    }
    else {
        std::cout << "initialized CUDA" << std::endl;
    }

    CudaStream s(true);
    CudaStream sdefault(false);
    std::cout << "stream is " << (s.is_default_stream() ? "" : "not") << " the default stream : " << std::endl;

    {
        const size_t N=128*1024*1024;
        unsigned int block_dim = 128;
        unsigned int grid_dim = N/block_dim + (N%block_dim ? 1 : 0);
        grid_dim = grid_dim > 1024 ? 1024 : grid_dim;
        dim3 block(block_dim);
        dim3 grid(grid_dim);
        std::cout << "launch grid : " << block_dim << "*" << grid_dim << std::endl;

        double *a_d, *b_d, *c_d;
        hipMalloc(&a_d, N*sizeof(double));
        hipMalloc(&b_d, N*sizeof(double));
        hipMalloc(&c_d, N*sizeof(double));

        double *a_h = new double[N];
        double *b_h = new double[N];
        double *c_h = new double[N];

        for(size_t i=0; i<N; ++i) {
            a_h[i] = 0.;
            b_h[i] = 1.;
            c_h[i] = 1.;
        }
        hipMemcpy(a_d, a_h, N*sizeof(double),  hipMemcpyHostToDevice);
        hipMemcpy(b_d, b_h, N*sizeof(double),  hipMemcpyHostToDevice);
        hipMemcpy(c_d, c_h, N*sizeof(double),  hipMemcpyHostToDevice);

        CudaEvent e;

        double time_init = omp_get_wtime();

        sum<<<block, grid, 0, s.stream()>>>(a_d, b_d, c_d, N);

        s.insert_event(e);

        double time_before_wait = omp_get_wtime();
        s.wait_on_event(e);
        double time_after_wait  = omp_get_wtime();
        e.wait();
        std::cout << "took " << time_before_wait-time_init << " " << time_after_wait-time_init << std::endl;

        hipMemcpy(a_h, a_d, N*sizeof(double), hipMemcpyDeviceToHost);
        size_t limit = 256;
        limit = N>limit ? limit : N;
        for(size_t i=N-limit; i<N; ++i)
            std::cout << a_h[i] << ((i+1)%block_dim ? " " : " | ");
        std::cout <<  std::endl;
    }

    return 0;
}

